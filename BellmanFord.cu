#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h> 



#define MAX_WEIGHT 100  // Maximum weight for an edge
#define MAX_LINE_LENGTH 1024  

typedef struct {
  int src; // source vertex of the edge
  int dest;  // destination vertex of the edge
  int weight;  // weight of the edge
} Edge;

typedef struct {
  int numVertices;
  int numEdges;
  Edge** edges;  // array of edges
} Graph;

// function to create an empty graph
Graph* createGraph(int numVertices, int numEdges) {
	//creating the graph
	Graph* graph = (Graph*)malloc(sizeof(Graph));
	graph->numVertices = numVertices;
	graph->numEdges = numEdges;

	// Creating a list of edges
	graph->edges = (Edge**)malloc(numEdges * sizeof(Edge));

	return graph;
}

// function to add and edge to the graph
void addEdge(Graph* graph, int numEdge, int src, int dest, int weight){
	// Allocate the new edge
	graph->edges[numEdge] = (Edge*)malloc(sizeof(Edge));

	Edge* newEdge = graph->edges[numEdge];
	// insert the information of the edge
	newEdge->src = src;
	newEdge->dest = dest;
	newEdge->weight = weight;
}

// function to print a graph to a file
void printGraph(Graph* graph, FILE *output_file){
	int numVertices = graph->numVertices;
	int numEdges = graph->numEdges;

	fprintf(output_file, "%d\n", numVertices);
	fprintf(output_file, "%d\n", numEdges);
	
	printf("numVertices: %d\n", numVertices);
	printf("numEdges: %d\n\n", numEdges);

	// for each vertex
	for (int v=0; v<numVertices; v++) {
		fprintf(output_file, "%d:", v);
		// printf("%d:", v);

		// for each edges
		for (int e=0; e<numEdges; e++){
			Edge* edge = graph->edges[e];
			if (edge->src == v){
				fprintf(output_file, "%d ", edge->dest);
				// printf("%d ", edge->dest);
				fprintf(output_file, "%d,", edge->weight);
				// printf("%d,", edge->weight);
			}
		}
		fprintf(output_file, "\n");
		//printf("\n");  
	}
}

int readNumber(FILE *file){
	/*
	given a file, read a number until there is a \n
	it is used to read the number of verteces and edges
	*/
	char c;
	char *string;
	int n = 0;
	int res;

	string = (char*) malloc(sizeof(char) * MAX_LINE_LENGTH);
	
	// read the character until end of line
	c = (char) fgetc(file);
	while (c != '\n'){
		string[n] = c;
		n++;
		c = (char) fgetc(file);
	}
	string[n] = '\0';
	res = atoi(string);
	
	// free the memory and return
	free(string);
	return res;
}

Graph* readFile(char *fileName){
	/*
	Given a file containing info, return the graph structure
	*/
	FILE *file = fopen(fileName, "r");
	char *string;
	int n = 0;
    
	int c1; // character (int)
	char c2; // character (char)

	int numVertices;
	int numEdges;

	int srcVertex;
	int destVertex;
	int weightEdge;

	int count = 0;

    if (file == NULL){
        return NULL; //could not open file
	}

	// printf("Reading the number of vertices\n");
	numVertices = readNumber(file);
	// printf("Reading the number of edges\n");
	numEdges = readNumber(file);

	// creating an empty graph
	// printf("Creating an empty graph\n");
	Graph* graph = createGraph(numVertices, numEdges);

	string = (char*) malloc(sizeof(char) * MAX_LINE_LENGTH);

	c1 = fgetc(file);
	while (c1 != EOF){
		c2 = (char) c1;

		//read the src vertex	
		while (c2 != ':'){
			string[n++] = c2;
			c1 = fgetc(file);
			c2 = (char) c1;
		}
		string[n] = '\0';
		srcVertex = atoi(string);
			
		n = 0;
		// read all the destinations
		c1 = fgetc(file);
		c2 = (char) c1;

		while (c2 != '\n'){

			// read the destVertex
			while (c2 != ' '){
				string[n++] = c2;
				c1 = fgetc(file);
				c2 = (char) c1;
			}
			string[n] = '\0';
			destVertex = atoi(string);

			// read the weight of the edge
			n = 0;
			c1 = fgetc(file);
			c2 = (char) c1;
			while (c2 != ','){
				string[n++] = c2;
				c1 = fgetc(file);
				c2 = (char) c1;
			}
			string[n] = '\0';
			weightEdge = atoi(string);

			// add the edge to the graph
			addEdge(graph, count++, srcVertex, destVertex, weightEdge);

			// restarting the loop to search other edges with the same source
			n = 0;
			c1 = fgetc(file);
			c2 = (char) c1;
		}
		c1 = fgetc(file);
    }

	free(string);  
	fclose(file);   

	return graph;
}

void printArray(int* array, int n){
	for (int i=0; i<n; i++){
		printf("%d: %d\n", i, array[i]);
	}
}













#define BLKDIM 256

__device__ long long pack(int dist, int pred) {
    return ((long long)dist << 32) | (pred & 0xFFFFFFFF);
}

__device__ __host__ int unpackDist(long long val) {
    return (int)(val >> 32);
}

__device__ __host__ int unpackPred(long long val) {
    return (int)(val & 0xFFFFFFFF);
}


__global__ void step1_init(long long* dist_pred, int numVertices, int sourceVertex){
	const int idx = blockIdx.x * BLKDIM + threadIdx.x;

	if (idx < numVertices) {
		// Initialize the distance to all vertices to infinity and predecessor to -1
		dist_pred[idx] = pack(INT_MAX, -1);
	}

	// The distance from the source to itself is zero
	if (idx == sourceVertex){
		dist_pred[idx] = pack(0, -1);
	}
}

__global__ void relaxEdges(int* srcs, int* dests, int* weights, long long* dist_pred, int numEdges){
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLKDIM + tid;

	if (idx < numEdges){
		int src = srcs[idx];
		// Extract distance from the source
		int dist_src = unpackDist(dist_pred[src]);

		if (dist_src != INT_MAX) {
			int dest = dests[idx];
			int weight = weights[idx];
			int new_dist = dist_src + weight;

			// Create the long long variable
			long long new_dist_pred = pack(new_dist, src);

			if (new_dist < unpackDist(dist_pred[dest])) {
				// Update the array
				atomicMin(&dist_pred[dest], new_dist_pred);
			}
		}
	}
}

__global__ void check_negative_cycle(int* srcs, int* dests, int* weights, long long* dist_pred, int numEdges, int* negative_cycles){
	__shared__ int local_res[BLKDIM];
	const int idx = blockIdx.x * BLKDIM + threadIdx.x;
	const int tid = threadIdx.x;

	if (idx < numEdges){ 
		int src = srcs[idx];
		int dest = dests[idx];
		int weight = weights[idx];
		int dist_src = unpackDist(dist_pred[src]);

		if (dist_src != INT_MAX && dist_src + weight < unpackDist(dist_pred[dest])) {
			local_res[tid] = 1;  
		}
		else{
			local_res[tid] = 0;
		}
	}
	else{
		local_res[tid] = 0;
	}

	__syncthreads();

	// reduction inside the block
    for (int stride = BLKDIM / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            local_res[tid] += local_res[tid + stride];
        }
        __syncthreads();
    }

	// saving the final value
	negative_cycles[blockIdx.x] = local_res[0];
}

void cudaParallelBellmanFord(Graph* graph, int* srcs, int* dests, int* weights, int sourceVertex){
	int V = graph->numVertices;
	int E = graph->numEdges;

	const size_t size_V = V * sizeof(long long);
	const size_t size_E = E * sizeof(int);

	// initializing the results of the algorithm
	long long* dist_pred; 
	long long* d_dist_pred;
	dist_pred = (long long*) malloc(size_V);
	hipMalloc((void **)&d_dist_pred, size_V);

	int how_many_blocks_V = (V + BLKDIM - 1) / BLKDIM;
	int how_many_blocks_E = (E + BLKDIM - 1) / BLKDIM;

	// initializing the copies of the edge information on the device	
	int* d_srcs; 
	int* d_dests;
	int* d_weights;
	hipMalloc((void **)&d_srcs, size_E);
	hipMalloc((void **)&d_dests, size_E);
	hipMalloc((void **)&d_weights, size_E);
	hipMemcpy(d_srcs, srcs, size_E, hipMemcpyHostToDevice);
	hipMemcpy(d_dests, dests, size_E, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, size_E, hipMemcpyHostToDevice);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Error: %s launching kernel\n", hipGetErrorString(status));
		exit(-1);
	} 

	// Step 1: initialize graph
	step1_init<<<how_many_blocks_V, BLKDIM>>>(d_dist_pred, V, sourceVertex);
	status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Error: %s launching kernel\n", hipGetErrorString(status));
		exit(-1);
	} 

	// Step 2: relax edges repeatedly
	for (int v = 0; v < V - 1; v++) {
		relaxEdges <<<how_many_blocks_E, BLKDIM>>> (d_srcs, d_dests, d_weights, d_dist_pred, E);
		hipDeviceSynchronize();
		
		status = hipGetLastError();
		if (status != hipSuccess) {
			printf("Error: %s launching kernel\n", hipGetErrorString(status));
			exit(-1);
		} 
	}
	
	// Step 3: check for negative-weight cycles
	int* negative_cycles;
	int* d_negative_cycles;
	negative_cycles = (int*)malloc(how_many_blocks_E * sizeof(int));
	hipMalloc((void **)&d_negative_cycles, how_many_blocks_E * sizeof(int));

	check_negative_cycle <<<how_many_blocks_E, BLKDIM>>> (d_srcs, d_dests, d_weights, d_dist_pred, E, d_negative_cycles);
	status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Error: %s launching kernel\n", hipGetErrorString(status));
		exit(-1);
	} 
	// Copy back the array containing the result of the computation
	hipMemcpy(negative_cycles, d_negative_cycles, how_many_blocks_E * sizeof(int), hipMemcpyDeviceToHost);

	// reduction of the values returned by each block
	int negative_cycle = 0;
	for (int num_block = 0; num_block < how_many_blocks_E; num_block++) {
		negative_cycle += negative_cycles[num_block];
	}
	if (negative_cycle > 0) {
		printf("GRAPH CONTAINS A NEGATIVE-WEIGHT CYCLE\n");
	}

	// copy back the results
	hipMemcpy(dist_pred, d_dist_pred, size_V, hipMemcpyDeviceToHost);

	int* dist = (int*)malloc(V * sizeof(int));
	int* predecessors = (int*)malloc(V * sizeof(int));
	for (int i = 0; i < V; i++) {
		dist[i] = unpackDist(dist_pred[i]);
		predecessors[i] = unpackPred(dist_pred[i]);
	}

	hipFree(d_dist_pred);
	hipFree(d_srcs);
	hipFree(d_dests);
	hipFree(d_weights);
	hipFree(d_negative_cycles);

    // printf("Distances (from: distance):\n");
	// printArray(dist, V);
	// printf("Predecessors (of: predecessor):\n");
	// printArray(predecessors, V);

	free(dist_pred);
	free(negative_cycles);
	free(dist);
	free(predecessors);

	return;
}

int main(int argc, char *argv[]) {
	
	if (argc != 2) {
		printf("Usage: %s <input_file>\n", argv[0]);
		return 1;
	}

	Graph* graph = readFile(argv[1]);
	
	Edge** edges = graph->edges;
	int numEdges = graph->numEdges;

	int* srcs = (int*)malloc(numEdges * sizeof(int));
	int* dests = (int*)malloc(numEdges * sizeof(int));
	int* weights = (int*)malloc(numEdges * sizeof(int));

	for (int i = 0; i < numEdges; i++){
		srcs[i] = edges[i]->src;
		dests[i] = edges[i]->dest;
		weights[i] = edges[i]->weight;
	}

	float elapsed_time = -1;
	hipEvent_t tstart, tstop;

	hipEventCreate(&tstart);
	hipEventCreate(&tstop);
	hipEventRecord(tstart, 0);

	cudaParallelBellmanFord(graph, srcs, dests, weights, 0);
	
	hipEventRecord(tstop, 0);
	hipEventSynchronize(tstop);
	hipEventElapsedTime(&elapsed_time, tstart, tstop);   
	
	elapsed_time /= 1000;
	printf("Cuda Elapsed time: %f s\n", elapsed_time);

	hipEventDestroy(tstart);
	hipEventDestroy(tstop);

	return 0;
}
